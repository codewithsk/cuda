/* objective
 * 	c = A*b  // A[m][n] is a matrix, b[n] and c[m] are vectors
 * compile: nvcc --gpu-architecture=compute_60 --gpu-code=sm_60 -O3 matvec.cu -o matvec
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>


#define EC(ans) { chkerr((ans), __FILE__, __LINE__); }
inline void chkerr(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) << " File: " << file << " Line: " << line << '\n';
        exit(-1);
    }
}

void init (double *mat, double *vec, int m , int n)
{
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            mat[i * n + j] = i * m + j;
        }
    }

    for (int j = 0; j < n; ++j)
    {
        vec[j] = j;
    }
}


void matvec_h(double* mat, double*vec_in , double* vec_out, int m, int n)
{
    for (int i = 0; i < m; ++i)
    {
        vec_out[i] = 0;

        for (int j = 0; j < n; ++j)
        {
            vec_out[i] += mat[i * n + j] * vec_in[j];
        }
    }
}

__global__ void matvec_d(double* mat, double*vec_in , double* vec_out, int m, int n)
{
    /// complete code
}

void validate (double *host, double *gpu, int m)
{

    for (int i = 0; i < m; ++i)
    {
        if(std::abs(host[i] - gpu[i]) > 1e-3)
        {
            std::cerr << "possible error at position " << i << " host: " << host[i] << " device " << gpu[i] << '\n';
        }
    }
}


int main(int argc, char *argv[])
{
    if(argc < 2)
    {
        std::cerr << "Usage: ./matvec M N\n";
        exit(-1);
    }

    int m = std::atoi(argv[1]);
    int n = std::atoi(argv[2]);

    /* Host alloc */
    double *mat = (double*) malloc (m * n * sizeof(double));
    double *vec_in = (double*) malloc (n * sizeof(double));
    double *vec_out = (double*) malloc (m * sizeof(double));

    /* Device alloc */
    /// complete code

    /* Initialize host memory*/
    init(mat, vec_in, m, n);

    /* host compute */
    matvec_h(mat, vec_in, vec_out, m, n);


    /* Copy from host to device */
    /// complete code


    /* call gpu kernel */
    /// complete code

    /* Copy from device to host */
    /// complete code


    /* host vs device validation */
    /// REPLACE one vec_out with the result array that you moved from device to host
    validate(vec_out, vec_out, m);


    /* be clean */
    free(mat);
    free(vec_in);
    free(vec_out);

    /// add code to free gpu memory


    return 0;
}




