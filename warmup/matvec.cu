/* objective
 * 	c = A*b  // A[m][n] is a matrix, b[n] and c[m] are vectors
 * compile: nvcc --gpu-architecture=compute_60 --gpu-code=sm_60 -O3 matvec.cu -o matvec
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>


#define EC(ans) { chkerr((ans), __FILE__, __LINE__); }
inline void chkerr(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) << " File: " << file << " Line: " << line << '\n';
        exit(-1);
    }
}

void init (double *mat, double *vec, int m , int n)
{
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            mat[i * n + j] = i * m + j;
        }
    }

    for (int j = 0; j < n; ++j)
    {
        vec[j] = 1;
    }
}


void matvec_h(double* mat, double*vec_in , double* vec_out, int m, int n)
{
    for (int i = 0; i < m; ++i)
    {
        vec_out[i] = 0;

        for (int j = 0; j < n; ++j)
        {
            vec_out[i] += mat[i * n + j] * vec_in[j];
        }
    }
}

__global__ void matvec_d(double* mat, double*vec_in , double* vec_out, int m, int n)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    
    if(id<n){
        double result = 0;
        #pragma unroll 4 
	for(int i=0;i<n;i++)
        	result+=mat[id*n+i] * vec_in[i];
        vec_out[id] = result;
    }


}

void validate (double *host, double *gpu, int m)
{

    for (int i = 0; i < m; ++i)
    {
        if(std::abs(host[i] - gpu[i]) > 1e-3)
        {
            std::cerr << "possible error at position " << i << " host: " << host[i] << " device " << gpu[i] << '\n';
        }
    }
}


int main(int argc, char *argv[])
{
    if(argc < 2)
    {
        std::cerr << "Usage: ./matvec M N\n";
        exit(-1);
    }

    int m = std::atoi(argv[1]);
    int n = std::atoi(argv[2]);

    /* Host alloc */
    double *mat = (double*) malloc (m * n * sizeof(double));
    double *vec_in = (double*) malloc (n * sizeof(double));
    double *vec_out = (double*) malloc (m * sizeof(double));
    double *vec_result = (double*) malloc (m * sizeof(double));

    /* Device alloc */
    double *d_mat, *d_vec_in, *d_vec_out;
    hipMalloc(&d_mat, m*n*sizeof(double));
    hipMalloc(&d_vec_in, n*sizeof(double));
    hipMalloc(&d_vec_out, m*sizeof(double));

    /* Initialize host memory*/
    init(mat, vec_in, m, n);

    /* host compute */
    matvec_h(mat, vec_in, vec_out, m, n);


    /* Copy from host to device */
    hipMemcpy(d_mat, mat, m*n*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_vec_in, vec_in, n*sizeof(double), hipMemcpyHostToDevice);

    /* call gpu kernel */
    dim3 threads(128);
    dim3 grid((int)ceil(m/128.00));    
    matvec_d<<<grid, threads>>>(d_mat, d_vec_in, d_vec_out,m,n);

    /* Copy from device to host */
    hipMemcpy(vec_result, d_vec_out,m*sizeof(double), hipMemcpyDeviceToHost); 


    /* host vs device validation */
    /// REPLACE one vec_out with the result array that you moved from device to host
    validate(vec_out, vec_result, m);


    /* be clean */
    free(mat);
    free(vec_in);
    free(vec_out);

    hipFree(d_mat);
    hipFree(d_vec_out);
    hipFree(d_vec_in);

    return 0;
}




